#include "hip/hip_runtime.h"
#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <glm/glm.hpp>

__global__ void swapFramesKernel(int boidsCount, glm::vec2 *positions,  glm::vec2 *velocities, glm::vec2 *newPositions, glm::vec2* newVelocities)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid < boidsCount)
    {
        positions[gid] = newPositions[gid];
        velocities[gid] = newVelocities[gid];
    }
}

void swapFrames(int boidsCount, glm::vec2 *positions, glm::vec2 *velocities, glm::vec2* newPositions, glm::vec2* newVelocities)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = boidsCount / threadsPerBlock + 1;

    swapFramesKernel<<<blocksPerGrid, threadsPerBlock>>>(boidsCount, positions, velocities, newPositions, newVelocities);
    hipDeviceSynchronize();
}


void sendData(int boidsCount, glm::vec2* host_positions, glm::vec2* host_velocities, glm::vec2* host_newPositions, glm::vec2* host_newVelocities)
{


    // hipMemcpy(device_newPositions, host_newPositions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
    // hipMemcpy(device_newVelocities, host_newVelocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);

    // swapFrames(boidsCount, device_positions, device_velocities, device_newPositions, device_newVelocities);

    // hipMemcpy(host_positions, device_positions, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    // hipMemcpy(host_velocities, device_velocities, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);

    // hipFree(device_positions);
    // hipFree(device_velocities);
    // hipFree(device_newPositions);
    // hipFree(device_newVelocities);
}


void allocateDataOnGPU(int boidsCount, glm::vec2*& device_positions, glm::vec2*& device_velocities, glm::vec2*& device_newPositions, glm::vec2*& device_newVelocities)
{
    // error handling
    hipMalloc((void**)&device_positions, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_velocities, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_newPositions, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_newVelocities, boidsCount * sizeof(glm::vec2));
}

void freeDataOnGPU(glm::vec2* device_positions, glm::vec2* device_velocities, glm::vec2* device_newPositions, glm::vec2* device_newVelocities)
{
    hipFree(device_positions);
    hipFree(device_velocities);
    hipFree(device_newPositions);
    hipFree(device_newVelocities);
}

void sendDataToGPU(int boidsCount, glm::vec2* device_newPositions, glm::vec2* device_newVelocities, glm::vec2* host_newPositions, glm::vec2* host_newVelocities)
{
    hipMemcpy(device_newPositions, host_newPositions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
    hipMemcpy(device_newVelocities, host_newVelocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
}

void getDataFromGPU(int boidsCount, glm::vec2* device_positions, glm::vec2* device_velocities, glm::vec2* host_positions, glm::vec2* host_velocities)
{
    hipMemcpy(host_positions, device_positions, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    hipMemcpy(host_velocities, device_velocities, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
}

// void sendDataToGPUv2(int boidsCount, glm::vec2* device_newPositions, glm::vec2* device_newVelocities, glm::vec2* host_newPositions, glm::vec2* host_newVelocities)
// {
//     hipMemcpy(device_newPositions, host_newPositions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
//     hipMemcpy(device_newVelocities, host_newVelocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
// }


void sendData2(int boidsCount, glm::vec2* host_positions, glm::vec2* host_velocities, glm::vec2* host_newPositions, glm::vec2* host_newVelocities)
{
    glm::vec2* device_positions;
    glm::vec2* device_velocities;

    glm::vec2* device_newPositions;
    glm::vec2* device_newVelocities;

    hipMalloc((void**)&device_positions, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_velocities, boidsCount * sizeof(glm::vec2));

    hipMalloc((void**)&device_newPositions, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_newVelocities, boidsCount * sizeof(glm::vec2));

    hipMemcpy(device_newPositions, host_newPositions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
    hipMemcpy(device_newVelocities, host_newVelocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);

    swapFrames(boidsCount, device_positions, device_velocities, device_newPositions, device_newVelocities);

    hipMemcpy(host_positions, device_positions, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    hipMemcpy(host_velocities, device_velocities, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);

    hipFree(device_positions);
    hipFree(device_velocities);
    hipFree(device_newPositions);
    hipFree(device_newVelocities);
}
