#include "hip/hip_runtime.h"
#define screenWidth 1000
#define screenHeight 1000

#define NUMBER_OF_SEGMENTS 50
#define RADIUS 10
#define START_X 500
#define START_Y 500

#define BORDER_FORCE 10
#define PERCEPTION 50
#define SLOWING_DISTANCE 100

#define MIN_SPEED 0.0f
#define MAX_SPEED 4.0f
#define MAX_FORCE 1.0f

#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <glm/glm.hpp>

#include "Grid.cuh"

namespace cuda_functions_grid
{
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
    {
        if (code != hipSuccess)
        {
            std::cerr << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
            if (abort)
                exit(code);
        }
    }
    __device__ glm::vec2 limit(glm::vec2 v, float l)
    {
        if (glm::length(v) > l)
        {
            if (length(v) > 0)
                v = glm::normalize(v) * l;
        }
        return v;
    }

    __device__ bool checkNeighbour(int gid, int neighIndex, BoidData& boidData)
    {
        if (gid == neighIndex)
            return false;
        if (fabs(glm::length(boidData.device_positions[gid] - boidData.device_positions[neighIndex])) <= PERCEPTION)
            return true;
        return false;
    }

    __device__ bool checkNeighbourGrid(glm::vec2 boidPosition, glm::vec2 neighPosition, BoidData& boidData)
    {
        if(boidPosition.x == neighPosition.x && boidPosition.y == neighPosition.y)
            return false;

        if (fabs(glm::length(boidPosition - neighPosition)) <= PERCEPTION)
            return true;
        return false;
    }

    __device__ void antiBorderCollisionThrough(int gid,  BoidData& boidData)
    {
        glm::vec2 &position = boidData.device_newPositions[gid];

        if (position.x < RADIUS)
            position.x = screenWidth - RADIUS;

        if (position.x + RADIUS > screenWidth)
            position.x = RADIUS;

        if (position.y < RADIUS)
            position.y = screenHeight - RADIUS;

        if (position.y + RADIUS > screenHeight)
            position.y = RADIUS;
    }

    __device__ glm::vec2 steeringForce(glm::vec2 target, glm::vec2 velocity)
    {
        // limit, normalize should prepared for the device !!!!!!
        glm::vec2 targetOffset = target;

        glm::vec2 desiredVelocity = {0, 0};
        glm::vec2 steeringForce = {0, 0};

        if (length(targetOffset) > 0)
        {
            desiredVelocity = normalize(targetOffset) * MAX_SPEED;
            glm::vec2 steeringVelocity = desiredVelocity - velocity;
            steeringForce = limit(steeringVelocity, MAX_FORCE);
        }
        return steeringForce;
    }

    __device__ glm::vec2 alignmentForceGrid(int gid, int boidsCount,  BoidData& boidData)
    {
        glm::vec2 target = glm::vec2(0, 0);
        int neighsCount = 0;

        int cell = grid::pixels2Cell(boidData.device_positions[gid].x, boidData.device_positions[gid].y, boidData.params);
        
        for(int i = boidData.device_gridCellStart[cell]; i <= boidData.device_gridCellEnd[cell]; i++)
        { 
            if(!checkNeighbourGrid(boidData.device_positions[gid], boidData.device_positionsSorted[i], boidData))
                continue;
            target += boidData.device_velocitiesSorted[i];
            neighsCount++;
        }

        int neighCells[9];
        grid::getAdjacentCells(cell, neighCells, boidData.params);

        for(int neighCell: neighCells)
        {
            if(neighCell == -1)
                continue;
            for(int i = boidData.device_gridCellStart[neighCell]; i <= boidData.device_gridCellEnd[neighCell]; i++)
            {
                if(!checkNeighbourGrid(boidData.device_positions[gid], boidData.device_positionsSorted[i], boidData))
                    continue;
                target += boidData.device_velocitiesSorted[i];
                neighsCount++;
            }
        }
        // for (int i = 0; i < boidsCount; i++)
        // {
        //     if (checkNeighbour(gid, i, boidData))
        //     {
        //         target += boidData.device_velocities[i];
        //         neighsCount++;
        //     }
        // }
        if (neighsCount > 0)
            target /= neighsCount;
        else
            target = boidData.device_velocities[gid];
            // target = boidData.device_velocities[gid];

        return steeringForce(target, boidData.device_velocities[gid]);
    }

    __device__ glm::vec2 alignmentForce(int gid, int boidsCount,  BoidData& boidData)
    {
        // consider saving result in alignmentForce
        glm::vec2 target = glm::vec2(0, 0);
        int neighsCount = 0;

        for (int i = 0; i < boidsCount; i++)
        {
            if (checkNeighbour(gid, i, boidData))
            {
                target += boidData.device_velocities[i];
                neighsCount++;
            }
        }
        if (neighsCount > 0)
            target /= neighsCount;
        else
            target = boidData.device_velocities[gid];

        return steeringForce(target, boidData.device_velocities[gid]);
    }

    __device__ glm::vec2 cohesionForceGrid(int gid, int boidsCount, BoidData& boidData)
    {
        glm::vec2 target = glm::vec2(0, 0);
        int neighsCount = 0;

        int cell = grid::pixels2Cell(boidData.device_positions[gid].x, boidData.device_positions[gid].y, boidData.params);
        
        for(int i = boidData.device_gridCellStart[cell]; i <= boidData.device_gridCellEnd[cell]; i++)
        { 
            if(!checkNeighbourGrid(boidData.device_positions[gid], boidData.device_positionsSorted[i], boidData))
                continue;
            target += boidData.device_positionsSorted[i];
            neighsCount++;
        }

        int neighCells[9];
        grid::getAdjacentCells(cell, neighCells, boidData.params);

        for(int neighCell: neighCells)
        {
            if(neighCell == -1)
                continue;
            for(int i = boidData.device_gridCellStart[neighCell]; i <= boidData.device_gridCellEnd[neighCell]; i++)
            {
                if(!checkNeighbourGrid(boidData.device_positions[gid], boidData.device_positionsSorted[i], boidData))
                    continue;
                target += boidData.device_positionsSorted[i];
                neighsCount++;
            }
        }

        if (neighsCount > 0)
            target /= neighsCount;
        else
            target = boidData.device_positions[gid];

        return steeringForce(target - boidData.device_positions[gid], boidData.device_velocities[gid]);
    }

    __device__ glm::vec2 cohesionForce(int gid, int boidsCount, BoidData& boidData)
    {
        glm::vec2 target = glm::vec2(0, 0);
        int neighsCount = 0;

        for (int i = 0; i < boidsCount; i++)
        {
            if (checkNeighbour(gid, i, boidData))
            {
                target += boidData.device_positions[i];
                neighsCount++;
            }
        }
        if (neighsCount > 0)
            target /= neighsCount;
        else
            target = boidData.device_positions[gid];

        return steeringForce(target - boidData.device_positions[gid], boidData.device_velocities[gid]);
    }

    __device__ glm::vec2 separationForceGrid(int gid, int boidsCount, BoidData& boidData)
    {
        // review force computation
        glm::vec2 target = glm::vec2(0, 0);
        int neighsCount = 0;

        int cell = grid::pixels2Cell(boidData.device_positions[gid].x, boidData.device_positions[gid].y, boidData.params);
        
        for(int i = boidData.device_gridCellStart[cell]; i <= boidData.device_gridCellEnd[cell]; i++)
        { 
            if(!checkNeighbourGrid(boidData.device_positions[gid], boidData.device_positionsSorted[i], boidData))
                continue;
            glm::vec2 offset = boidData.device_positions[gid] - boidData.device_positionsSorted[i];
                if (length(offset) == 0)
                    continue;

                // value = normalize(offset) * (1 / length(offset));
                glm::vec2 value = offset * (1 / length(offset));
                target += value;
                neighsCount++;
        }

        int neighCells[9];
        grid::getAdjacentCells(cell, neighCells, boidData.params);

        for(int neighCell: neighCells)
        {
            if(neighCell == -1)
                continue;
            
            for(int i = boidData.device_gridCellStart[neighCell]; i <= boidData.device_gridCellEnd[neighCell]; i++)
            {
                if(!checkNeighbourGrid(boidData.device_positions[gid], boidData.device_positionsSorted[i], boidData))
                    continue;
                glm::vec2 offset = boidData.device_positions[gid] - boidData.device_positionsSorted[i];
                if (length(offset) == 0)
                    continue;

                // value = normalize(offset) * (1 / length(offset));
                glm::vec2 value = offset * (1 / length(offset));
                target += value;
                neighsCount++;
            }
        }


        // for (int i = 0; i < boidsCount; i++)
        // {
        //     if (checkNeighbour(gid, i, boidData))
        //     {
        //         glm::vec2 offset = boidData.device_positions[gid] - boidData.device_positions[i];
        //         if (length(offset) == 0)
        //             continue;

        //         // value = normalize(offset) * (1 / length(offset));
        //         glm::vec2 value = offset * (1 / length(offset));
        //         target += value;
        //         neighsCount++;
        //     }
        // }

        if (neighsCount > 0)
            target /= neighsCount;
        else
            return glm::vec2(0, 0);

        return steeringForce(target, boidData.device_velocities[gid]);
    }


    __device__ glm::vec2 separationForce(int gid, int boidsCount, BoidData& boidData)
    {
        // review force computation
        glm::vec2 target = glm::vec2(0, 0);
        int neighsCount = 0;

        for (int i = 0; i < boidsCount; i++)
        {
            if (checkNeighbour(gid, i, boidData))
            {
                glm::vec2 offset = boidData.device_positions[gid] - boidData.device_positions[i];
                if (length(offset) == 0)
                    continue;

                // value = normalize(offset) * (1 / length(offset));
                glm::vec2 value = offset * (1 / length(offset));
                target += value;
                neighsCount++;
            }
        }

        if (neighsCount > 0)
            target /= neighsCount;
        else
            return glm::vec2(0, 0);

        return steeringForce(target, boidData.device_velocities[gid]);
    }

    __device__ glm::vec2 cursorDodgeForce(int gid, int boidsCount, BoidData& boidData)
    {

        glm::vec2 target = glm::vec2(0, 0);
                glm::vec2 offset = boidData.device_positions[gid] - glm::vec2(boidData.params.cursorX, boidData.params.cursorY);


                if (length(offset) == 0 || length(offset) > 100.0)
                    return glm::vec2(0, 0);

                glm::vec2 value = offset * (1 / length(offset));
                target += value;
        return steeringForce(target, boidData.device_velocities[gid]);
    }

    __device__ void applyForces(int gid, int boidsCount, BoidData& boidData)
    {
        boidData.device_accelerations[gid] *= 0;
        // boidData.device_accelerations[gid] += alignmentForce(gid, boidsCount, boidData);
        // boidData.device_accelerations[gid] += (cohesionForce(gid, boidsCount, boidData));
        // boidData.device_accelerations[gid] += (separationForce(gid, boidsCount, boidData));        
        
        boidData.device_accelerations[gid] += alignmentForceGrid(gid, boidsCount, boidData);
        boidData.device_accelerations[gid] += (cohesionForceGrid(gid, boidsCount, boidData));
        boidData.device_accelerations[gid] += (separationForceGrid(gid, boidsCount, boidData));
        boidData.device_accelerations[gid] += 5.0f * (cursorDodgeForce(gid, boidsCount, boidData));

        // // auto k1 = length(separationForce(gid, boidsCount, boidData));
        // // auto k2 = length(separationForceGrid(gid, boidsCount, boidData));        
        
        // auto k1 = length(alignmentForce(gid, boidsCount, boidData));
        // auto k2 = length(alignmentForceGrid(gid, boidsCount, boidData));
        // if(fabs(k1 - k2)> 0.0)
        // {
        //     printf("kur...\n");
        // }
        // else
        //     printf("ok\n");
    }

    __global__ void computeNextFrameKernel(int boidsCount, BoidData boidData)
    {
        int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < boidsCount)
        {
            applyForces(gid, boidsCount, boidData);
            boidData.device_newVelocities[gid] = boidData.device_velocities[gid] + boidData.device_accelerations[gid];

            boidData.device_newPositions[gid] = boidData.device_positions[gid] + boidData.device_velocities[gid];

            antiBorderCollisionThrough(gid, boidData);

            // translations[gid] = newPositions[gid] - START;
            boidData.device_translations[gid] = boidData.device_newPositions[gid] - glm::vec2(START_X, START_Y);
            // printf("boidData.device_translations[gid]: %f: %f\n", boidData.device_translations[gid].x, boidData.device_translations[gid].y);
        }
    }

    void computeNextFrame(int boidsCount, BoidData boidData)
    {
        // consider passing by reference

        // parameters rearranged!!!!!
        // int threadsPerBlock = 10;
        // int blocksPerGrid = boidsCount / threadsPerBlock + 1;
        // blocksPerGrid*=2;
        //  blocksPerGrid = 60;
        int threadsPerBlock = 128;
        int blocksPerGrid = (boidsCount + threadsPerBlock - 1) / threadsPerBlock;

        computeNextFrameKernel<<<blocksPerGrid, threadsPerBlock>>>(boidsCount, boidData);
        // hipDeviceSynchronize();
        gpuErrchk(hipGetLastError());

        // Check for errors on the CUDA device side after kernel execution
        gpuErrchk(hipDeviceSynchronize());
    }
    
    __global__ void swapFramesKernel(int boidsCount, glm::vec2 *positions, glm::vec2 *velocities, glm::vec2 *newPositions, glm::vec2 *newVelocities)
    {
        int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < boidsCount)
        {
            positions[gid] = newPositions[gid];
            velocities[gid] = newVelocities[gid];
        }
    }

    void swapFrames(int boidsCount, glm::vec2 *positions, glm::vec2 *velocities, glm::vec2 *newPositions, glm::vec2 *newVelocities)
    {
        // int threadsPerBlock = 10;
        // int blocksPerGrid = boidsCount / threadsPerBlock + 1;
        printf("fff");
        // blocksPerGrid*=2;
        int threadsPerBlock = 128;
        int blocksPerGrid = (boidsCount + threadsPerBlock - 1) / threadsPerBlock;

        swapFramesKernel<<<blocksPerGrid, threadsPerBlock>>>(boidsCount, positions, velocities, newPositions, newVelocities);
        // hipDeviceSynchronize();
        gpuErrchk(hipGetLastError());

        // Check for errors on the CUDA device side after kernel execution
        gpuErrchk(hipDeviceSynchronize());
    }

}