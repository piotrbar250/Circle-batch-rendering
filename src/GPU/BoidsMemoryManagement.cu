#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <glm/glm.hpp>

namespace cuda_functions
{
    void allocateDataOnGPU(int boidsCount, glm::vec2 *&device_positions, glm::vec2 *&device_velocities, glm::vec2 *&device_newPositions, glm::vec2 *&device_newVelocities, glm::vec2 *&device_accelerations, glm::vec2 *&device_translations)
    {
        // error handling
        hipMalloc((void **)&device_positions, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_velocities, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_newPositions, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_newVelocities, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_accelerations, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_translations, boidsCount * sizeof(glm::vec2));
    }

    void freeDataOnGPU(glm::vec2 *device_positions, glm::vec2 *device_velocities, glm::vec2 *device_newPositions, glm::vec2 *device_newVelocities, glm::vec2 *device_accelerations, glm::vec2 *device_translations)
    {
        hipFree(device_positions);
        hipFree(device_velocities);
        hipFree(device_newPositions);
        hipFree(device_newVelocities);
        hipFree(device_accelerations);
        hipFree(device_translations);
    }

    void sendDataToGPU(int boidsCount, glm::vec2 *device_positions, glm::vec2 *device_velocities, glm::vec2 *host_positions, glm::vec2 *host_velocities)
    {
        hipMemcpy(device_positions, host_positions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
        hipMemcpy(device_velocities, host_velocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
    }

    void getDataFromGPU(int boidsCount, glm::vec2 *device_translations, glm::vec2 *host_translations)
    {
        hipMemcpy(host_translations, device_translations, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    }

    void registerVBO(void** cuda_vbo_resource, unsigned int instanceVBO)
    {
        //unregister and cleanup
        hipGraphicsGLRegisterBuffer((hipGraphicsResource**)cuda_vbo_resource, instanceVBO, cudaGraphicsMapFlagsWriteDiscard);
    }

    glm::vec2* getMappedPointer(void **cuda_vbo_resource)
    {
        hipDeviceSynchronize();
        // hipGraphicsMapResources(1, (hipGraphicsResource**)cuda_vbo_resource, 0);
        if(cuda_vbo_resource == nullptr)
        {
            printf("oho\n");
        }
        if(*cuda_vbo_resource == nullptr)
        {
            printf("faken\n");
        }

        hipGraphicsResource** a = (hipGraphicsResource**)cuda_vbo_resource;
        printf("a\n");
        hipGraphicsResource* b = *a;
        printf("b\n");
        hipGraphicsResource_t* c = (hipGraphicsResource_t*)b;
        printf("c\n");
        // it works until this point

        hipGraphicsMapResources(1, c, 0);  // now it crashes

        // hipGraphicsMapResources(1, (hipGraphicsResource_t*)*cuda_vbo_resource, 0);
        printf("to sie wysypie\n");
        size_t num_bytes;
        glm::vec2* devPtr;
        hipGraphicsResourceGetMappedPointer((void**)&devPtr, &num_bytes, (hipGraphicsResource*)*cuda_vbo_resource);
        return devPtr;
    }

    void unmapResource(void **cuda_vbo_resource)
    {
        hipGraphicsUnmapResources(1, (hipGraphicsResource**)cuda_vbo_resource,0);
        // hipGraphicsUnmapResources(1, (hipGraphicsResource_t*)*cuda_vbo_resource,0);
    }
}