#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <glm/glm.hpp>

namespace cuda_functions
{
    void allocateDataOnGPU(int boidsCount, glm::vec2 *&device_positions, glm::vec2 *&device_velocities, glm::vec2 *&device_newPositions, glm::vec2 *&device_newVelocities, glm::vec2 *&device_accelerations, glm::vec2 *&device_translations)
    {
        // error handling
        hipMalloc((void **)&device_positions, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_velocities, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_newPositions, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_newVelocities, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_accelerations, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_translations, boidsCount * sizeof(glm::vec2));
    }

    void freeDataOnGPU(glm::vec2 *device_positions, glm::vec2 *device_velocities, glm::vec2 *device_newPositions, glm::vec2 *device_newVelocities, glm::vec2 *device_accelerations, glm::vec2 *device_translations)
    {
        hipFree(device_positions);
        hipFree(device_velocities);
        hipFree(device_newPositions);
        hipFree(device_newVelocities);
        hipFree(device_accelerations);
        hipFree(device_translations);
    }

    void sendDataToGPU(int boidsCount, glm::vec2 *device_positions, glm::vec2 *device_velocities, glm::vec2 *host_positions, glm::vec2 *host_velocities)
    {
        hipMemcpy(device_positions, host_positions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
        hipMemcpy(device_velocities, host_velocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
    }

    void getDataFromGPU(int boidsCount, glm::vec2 *device_translations, glm::vec2 *host_translations)
    {
        hipMemcpy(host_translations, device_translations, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    }

    void registerVBO(void** cuda_vbo_resource, unsigned int instanceVBO)
    {
        //unregister and cleanup
        hipGraphicsGLRegisterBuffer((hipGraphicsResource**)cuda_vbo_resource, instanceVBO, cudaGraphicsMapFlagsWriteDiscard);
    }

    glm::vec2* getMappedPointer(void **cuda_vbo_resource)
    {

        hipGraphicsMapResources(1, (hipGraphicsResource**)cuda_vbo_resource, 0);
        // printf("to sie wysypie\n");
        size_t num_bytes;
        glm::vec2* devPtr;
        hipGraphicsResourceGetMappedPointer((void**)&devPtr, &num_bytes, (hipGraphicsResource*)*cuda_vbo_resource);
        return devPtr;
    }

    void unmapResource(void **cuda_vbo_resource)
    {
        hipGraphicsUnmapResources(1, (hipGraphicsResource**)cuda_vbo_resource,0);
        // hipGraphicsUnmapResources(1, (hipGraphicsResource_t*)*cuda_vbo_resource,0);
    }
}