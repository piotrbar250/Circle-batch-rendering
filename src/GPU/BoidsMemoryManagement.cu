#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#include <stdio.h>
#include <glm/glm.hpp>
#include "Grid.cuh"

namespace cuda_functions
{
    void allocateDataOnGPU(int boidsCount, glm::vec2 *&device_positions, glm::vec2 *&device_velocities, glm::vec2 *&device_newPositions, glm::vec2 *&device_newVelocities, glm::vec2 *&device_accelerations, glm::vec2 *&device_translations)
    {
        hipMalloc((void **)&device_positions, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_velocities, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_newPositions, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_newVelocities, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_accelerations, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_translations, boidsCount * sizeof(glm::vec2));
       
    }

    void freeDataOnGPU(glm::vec2 *device_positions, glm::vec2 *device_velocities, glm::vec2 *device_newPositions, glm::vec2 *device_newVelocities, glm::vec2 *device_accelerations, glm::vec2 *device_translations)
    {
        hipFree(device_positions);
        hipFree(device_velocities);
        hipFree(device_newPositions);
        hipFree(device_newVelocities);
        hipFree(device_accelerations);
        hipFree(device_translations);
    }

    void sendDataToGPU(int boidsCount, glm::vec2 *device_positions, glm::vec2 *device_velocities, glm::vec2 *host_positions, glm::vec2 *host_velocities)
    {
        hipMemcpy(device_positions, host_positions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
        hipMemcpy(device_velocities, host_velocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
    }

    void getDataFromGPU(int boidsCount, glm::vec2 *device_translations, glm::vec2 *host_translations)
    {
        hipMemcpy(host_translations, device_translations, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    }

    void registerVBO(void** cuda_vbo_resource, unsigned int instanceVBO)
    {
        hipGraphicsGLRegisterBuffer((hipGraphicsResource**)cuda_vbo_resource, instanceVBO, cudaGraphicsMapFlagsWriteDiscard);
    }

    glm::vec2* getMappedPointer(void **cuda_vbo_resource)
    {

        hipGraphicsMapResources(1, (hipGraphicsResource**)cuda_vbo_resource, 0);
        size_t num_bytes;
        glm::vec2* devPtr;
        hipGraphicsResourceGetMappedPointer((void**)&devPtr, &num_bytes, (hipGraphicsResource*)*cuda_vbo_resource);
        return devPtr;
    }

    void unmapResource(void **cuda_vbo_resource)
    {
        hipGraphicsUnmapResources(1, (hipGraphicsResource**)cuda_vbo_resource,0);
    }

    void sendColorsToGPU(int boidsCount, int* device_colorIndex, int* host_colorIndex)
    {
        hipMemcpy(device_colorIndex, host_colorIndex, boidsCount * sizeof(int), hipMemcpyHostToDevice);
    }

}

namespace cuda_functions_grid
{
    void allocateGrid(int boidsCount, int cellCount, int *&device_gridCellIndex, int *&device_boidSequence, int *&device_gridCellStart, int *&device_gridCellEnd, glm::vec2*& device_positionsSorted, glm::vec2*& device_velocitiesSorted,int*& colorIndex ,int*& colorSorted)
    {
        hipMalloc((void**)&device_gridCellIndex, boidsCount*sizeof(int));
        hipMalloc((void**)&device_boidSequence, boidsCount*sizeof(int));
        hipMalloc((void**)&device_gridCellStart, cellCount*sizeof(int));
        hipMalloc((void**)&device_gridCellEnd, cellCount*sizeof(int));
        hipMalloc((void**)&device_positionsSorted, boidsCount*sizeof(glm::vec2));
        hipMalloc((void**)&device_velocitiesSorted, boidsCount*sizeof(glm::vec2));
        hipMalloc((void **)&colorIndex, boidsCount * sizeof(int));
        hipMalloc((void**)&colorSorted, boidsCount*sizeof(int));
    } 
}