#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <glm/glm.hpp>

namespace cuda_functions
{
    void allocateDataOnGPU(int boidsCount, glm::vec2 *&device_positions, glm::vec2 *&device_velocities, glm::vec2 *&device_newPositions, glm::vec2 *&device_newVelocities, glm::vec2 *&device_accelerations, glm::vec2 *&device_translations)
    {
        // error handling
        hipMalloc((void **)&device_positions, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_velocities, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_newPositions, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_newVelocities, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_accelerations, boidsCount * sizeof(glm::vec2));
        hipMalloc((void **)&device_translations, boidsCount * sizeof(glm::vec2));
    }

    void freeDataOnGPU(glm::vec2 *device_positions, glm::vec2 *device_velocities, glm::vec2 *device_newPositions, glm::vec2 *device_newVelocities, glm::vec2 *device_accelerations, glm::vec2 *device_translations)
    {
        hipFree(device_positions);
        hipFree(device_velocities);
        hipFree(device_newPositions);
        hipFree(device_newVelocities);
        hipFree(device_accelerations);
        hipFree(device_translations);
    }

    void sendDataToGPU(int boidsCount, glm::vec2 *device_positions, glm::vec2 *device_velocities, glm::vec2 *host_positions, glm::vec2 *host_velocities)
    {
        hipMemcpy(device_positions, host_positions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
        hipMemcpy(device_velocities, host_velocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
    }

    void getDataFromGPU(int boidsCount, glm::vec2 *device_translations, glm::vec2 *host_translations)
    {
        hipMemcpy(host_translations, device_translations, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    }
}