#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <glm/glm.hpp>


void allocateDataOnGPU(int boidsCount, glm::vec2*& device_positions, glm::vec2*& device_velocities, glm::vec2*& device_newPositions, glm::vec2*& device_newVelocities, glm::vec2*& device_accelerations, glm::vec2*& device_translations)
{
    // error handling
    hipMalloc((void**)&device_positions, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_velocities, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_newPositions, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_newVelocities, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_accelerations, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_translations, boidsCount * sizeof(glm::vec2));
}

void freeDataOnGPU(glm::vec2* device_positions, glm::vec2* device_velocities, glm::vec2* device_newPositions, glm::vec2* device_newVelocities, glm::vec2* device_accelerations, glm::vec2* device_translations)
{
    hipFree(device_positions);
    hipFree(device_velocities);
    hipFree(device_newPositions);
    hipFree(device_newVelocities);
    hipFree(device_accelerations);
    hipFree(device_translations);
}

void sendDataToGPUinit(int boidsCount, glm::vec2* device_positions, glm::vec2* device_velocities, glm::vec2* host_positions, glm::vec2* host_velocities)
{
    hipMemcpy(device_positions, host_positions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
    hipMemcpy(device_velocities, host_velocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
}

void getDataFromGPUinit(int boidsCount, glm::vec2* device_translations, glm::vec2* host_translations)
{
    hipMemcpy(host_translations, device_translations, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
}


void sendDataToGPU(int boidsCount, glm::vec2* device_newPositions, glm::vec2* device_newVelocities, glm::vec2* host_newPositions, glm::vec2* host_newVelocities)
{
    hipMemcpy(device_newPositions, host_newPositions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
    hipMemcpy(device_newVelocities, host_newVelocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
}

void getDataFromGPU(int boidsCount, glm::vec2* device_positions, glm::vec2* device_velocities, glm::vec2* host_positions, glm::vec2* host_velocities)
{
    hipMemcpy(host_positions, device_positions, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    hipMemcpy(host_velocities, device_velocities, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
}


void sendData2(int boidsCount, glm::vec2* host_positions, glm::vec2* host_velocities, glm::vec2* host_newPositions, glm::vec2* host_newVelocities)
{
    glm::vec2* device_positions;
    glm::vec2* device_velocities;

    glm::vec2* device_newPositions;
    glm::vec2* device_newVelocities;

    hipMalloc((void**)&device_positions, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_velocities, boidsCount * sizeof(glm::vec2));

    hipMalloc((void**)&device_newPositions, boidsCount * sizeof(glm::vec2));
    hipMalloc((void**)&device_newVelocities, boidsCount * sizeof(glm::vec2));

    hipMemcpy(device_newPositions, host_newPositions, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
    hipMemcpy(device_newVelocities, host_newVelocities, boidsCount * sizeof(glm::vec2), hipMemcpyHostToDevice);

    swapFrames(boidsCount, device_positions, device_velocities, device_newPositions, device_newVelocities);

    hipMemcpy(host_positions, device_positions, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    hipMemcpy(host_velocities, device_velocities, boidsCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);

    hipFree(device_positions);
    hipFree(device_velocities);
    hipFree(device_newPositions);
    hipFree(device_newVelocities);
}
