#include "hip/hip_runtime.h"
#include "cuda_dependencies.cu"

namespace cuda_functions_grid
{

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
    {
        if (code != hipSuccess)
        {
            std::cerr << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
            if (abort)
                exit(code);
        }
    }

    __global__ void initStartEndKernel(int cellCount, int* device_gridCellStart, int* device_gridCellEnd)
    {
         int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if(gid < cellCount)
        {
            device_gridCellStart[gid] = -1;
            device_gridCellEnd[gid] = -2;
        }
        // printf("cell %d, device_gridCellStart[cell] %d\n", gid, device_gridCellStart[gid]);
    }

    __global__ void boidCellKernel(int boidsCount, GridParams params, glm::vec2* positions, int* gridCellIndex)
    {
        int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if(gid < boidsCount)
        {
            gridCellIndex[gid] = grid::pixels2Cell(positions[gid].x, positions[gid].y, params);
        }
    }

    __global__ void cellStartEndKernel(int boidCount,  int* device_gridCellIndex, int* device_gridCellStart, int* device_gridCellEnd)
    {
        int gid = blockIdx.x * blockDim.x + threadIdx.x;
        // printf("jakie gidy: %d\n", gid);
        if(gid < boidCount)
        {   
            if(gid == 0 || device_gridCellIndex[gid] != device_gridCellIndex[gid-1])
            {
                device_gridCellStart[device_gridCellIndex[gid]] = gid;
            }
            if(gid == boidCount-1 || device_gridCellIndex[gid] != device_gridCellIndex[gid+1])
            {
                device_gridCellEnd[device_gridCellIndex[gid]] = gid;
            }
            // printf("gid %d, cell %d, device_gridCellStart[cell] %d\n", gid, device_gridCellIndex[gid], device_gridCellStart[device_gridCellIndex[gid]]);
        }
    }

    void computeGridCellIndex(int boidsCount, GridParams params, glm::vec2* device_positions, glm::vec2* device_velocities, int* device_gridCellIndex, int* device_gridCellStart, int* device_gridCellEnd, int* boidSequence, glm::vec2* device_positionsSorted, glm::vec2* device_velocitiesSorted)
    {
        int threadsPerBlock = 128;
        int blocksPerGrid = (boidsCount + threadsPerBlock - 1) / threadsPerBlock;
        boidCellKernel<<<blocksPerGrid, threadsPerBlock>>>(boidsCount, params, device_positions, device_gridCellIndex);
        // hipDeviceSynchronize();
        gpuErrchk(hipGetLastError());

        // Check for errors on the CUDA device side after kernel execution
        gpuErrchk(hipDeviceSynchronize());

        thrust::device_vector<int> sequence(boidsCount);
        thrust::sequence(thrust::device, sequence.begin(), sequence.end());
        thrust::sort_by_key(thrust::device, device_gridCellIndex, device_gridCellIndex+boidsCount, sequence.begin());

        // for(int i = 0; i < boidsCount; i++)
        // {
        //     printf("i: %d, device_gridCellIndex: %d\n", i, device_gridCellIndex[i]);
        // }

        // for(int i = 0; i < boidsCount; i++)
        // {
        //     printf("i: %d, sequence: %d\n", i, sequence[i]);
        // }

        thrust::device_vector<glm::vec2> device_positionsSortedVector(boidsCount);
        thrust::device_vector<glm::vec2> device_velocitiesSortedVector(boidsCount);

        thrust::device_ptr<glm::vec2> dev_ptr_positions(device_positions);
        thrust::gather(sequence.begin(), sequence.end(), dev_ptr_positions, device_positionsSortedVector.begin());

        thrust::device_ptr<glm::vec2> dev_ptr_velocities(device_velocities);
        thrust::gather(sequence.begin(), sequence.end(), dev_ptr_velocities, device_velocitiesSortedVector.begin());
// printf("hello\n");
        // thrust::gather(sequence.begin(), sequence.end(), device_positions, device_positionsSorted);
        // thrust::gather(sequence.begin(), sequence.end(), device_positionsSortedVector.begin(), device_velocitiesSortedVector.begin());
        // thrust::gather(device_velocities,device_velocities, device_positionsSortedVector.begin(), device_velocitiesSortedVector.begin());
        
        // thrust::gather(sequence.begin(), sequence.end(), device_velocities, device_velocitiesSorted);
        // hipMalloc(&device_positionsSorted, boidsCount * sizeof(glm::vec2));

        thrust::copy(device_positionsSortedVector.begin(), device_positionsSortedVector.end(), device_positionsSorted);
        thrust::copy(device_velocitiesSortedVector.begin(), device_velocitiesSortedVector.end(), device_velocitiesSorted);

        blocksPerGrid = (params.cellCount + threadsPerBlock - 1) / threadsPerBlock;

        // printf("cellCount: %d\n", params.cellCount);
        initStartEndKernel<<<blocksPerGrid, threadsPerBlock>>>(params.cellCount, device_gridCellStart, device_gridCellEnd);
        gpuErrchk(hipGetLastError());

        // Check for errors on the CUDA device side after kernel execution
        gpuErrchk(hipDeviceSynchronize());

        blocksPerGrid = (boidsCount + threadsPerBlock - 1) / threadsPerBlock;

        cellStartEndKernel<<<blocksPerGrid, threadsPerBlock>>>(boidsCount, device_gridCellIndex, device_gridCellStart, device_gridCellEnd);


        // for(int i = 0; i < params.cellCount; i++)
        // {
        //     printf("i: %d, device_gridCellStart: %d\n", i, device_gridCellStart[i]);
        // }

        // hipDeviceSynchronize();

        gpuErrchk(hipGetLastError());

        // Check for errors on the CUDA device side after kernel execution
        gpuErrchk(hipDeviceSynchronize());

        // printf("passed\n");
        // int N = boidsCount;

        // thrust::device_vector<glm::vec2> dev_pos(N);             // Positions of boids
        // thrust::device_vector<glm::vec2> dev_vel1(N);
        // thrust::device_vector<glm::vec2> sorted_pos(N);
        // thrust::device_vector<glm::vec2> sorted_vel1(N);


        // thrust::device_ptr<glm::vec2> dev_ptr_positions(device_positions);

        // thrust::gather(sequence.begin(), sequence.end(), dev_ptr_positions, device_positionsSortedVector.begin());
        // thrust::gather(sequence.begin(), sequence.end(), dev_vel1.begin(), sorted_vel1.begin());
    
    }
}